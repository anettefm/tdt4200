#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cutil.h>


__global__ void invert(unsigned char* image_h, unsigned char* image_d ){

	int id=blockIdx.x*blockDim.x+threadIdx.x;
	image_d[id]=~image_h[id];
}

