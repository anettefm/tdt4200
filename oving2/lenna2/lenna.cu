#include "hip/hip_runtime.h"
#include <iostream>
#include "lodepng.h"
#include <stdio.h>

__global__ void invert( unsigned char* image_d ){

        int id=blockIdx.x*blockDim.x+threadIdx.x;
        image_d[id] = ~image_d[id];
}


int main( int argc, char ** argv){
        hipEvent_t event1, event2, event3, event4;
        hipEventCreate(&event1);
        hipEventCreate(&event2);
        hipEventCreate(&event3);
        hipEventCreate(&event4);
        hipEventRecord(event3, 0);


  size_t pngsize;
  unsigned char *png;
  const char * filename = "lenna512x512_inv.png";
  /* Read in the image */
  lodepng_load_file(&png, &pngsize, filename);
 
  unsigned char *image;
  unsigned int width, height;
  /* Decode it into a RGB 8-bit per channel vector */
  unsigned int error = lodepng_decode24(&image, &width, &height, png, pngsize);

  /* Check if read and decode of .png went well */
  if(error != 0){
      std::cout << "error " << error << ": " << lodepng_error_text(error) << std::endl;
  }

  // Do work
        /*dim3 gridblock, threadBlock;
        gridBlock.x=4; gridBlock.y=4;
        threadBlock.x=5; threadBlock.y=5;
        invert<<<gridBlock, threadBlock>>>(image) */

        unsigned char* image_d;

        size_t size;
        size=height*width*3*sizeof(char);

        hipMalloc((void**) &image_d, size);
	
//	hipEvent_t event1, event2;
//	hipEventCreate(&event1);
//	hipEventCreate(&event2);
	
	hipEventRecord(event1, 0);	
	hipMemcpy(image_d, image, size, hipMemcpyHostToDevice);
        hipEventRecord(event2, 0);
	
	hipEventSynchronize(event2);
	
	float dt_ms;
	hipEventElapsedTime(&dt_ms, event1, event2);
	
	printf("%f\n", dt_ms);
	int threadBlock=1024;
	int gridBlock=3*512*512/threadBlock;

        invert<<<gridBlock, threadBlock>>>(image_d);

        hipMemcpy(image, image_d, size, hipMemcpyDeviceToHost);
  /* Save the result to a new .png file */
  lodepng_encode24_file("lenna512x512_orig.png", image , width,height);
  /*clean up */
        free(image);  hipFree(image_d);
	 hipEventRecord(event4, 0);
        hipEventSynchronize(event4);
        float dt_ms_tot;
        hipEventElapsedTime(&dt_ms_tot, event3, event4);
        printf("%f\n", dt_ms_tot);


  return 0;
}
