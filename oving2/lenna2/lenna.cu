#include "hip/hip_runtime.h"
#include <iostream>
#include "lodepng.h"

__global__ void invert(unsigned char* image_h, unsigned char* image_d ){

        int id=blockIdx.x*blockDim.x+threadIdx.x;
        image_d[id]=~image_h[id];
}


int main( int argc, char ** argv){

  size_t pngsize;
  unsigned char *png;
  const char * filename = "lenna512x512_inv.png";
  /* Read in the image */
  lodepng_load_file(&png, &pngsize, filename);
 
  unsigned char *image;
  unsigned int width, height;
  /* Decode it into a RGB 8-bit per channel vector */
  unsigned int error = lodepng_decode24(&image, &width, &height, png, pngsize);

  /* Check if read and decode of .png went well */
  if(error != 0){
      std::cout << "error " << error << ": " << lodepng_error_text(error) << std::endl;
  }

  // Do work
        /*dim3 gridblock, threadBlock;
        gridBlock.x=4; gridBlock.y=4;
        threadBlock.x=5; threadBlock.y=5;
        invert<<<gridBlock, threadBlock>>>(image) */

        unigned char* image_d;

        size_t size;
        size=high*width*3*sizeof(char);

        hipMalloc((void**) &image_d, size);
        hipMemcpy(image, image_d, size, hipMemcpyHostToDevice);
        int gridBlock;
        int threadBlock;

        invert<<gridBlock, threadBlock>>(image, image_d);

        hipMemcpy(image_d, image, hipMemcpyDeviceToHost);

  /* Save the result to a new .png file */
  lodepng_encode24_file("lenna512x512_orig.png", image , width,height);
  /*clean up */
        free(image);  hipFree(image_d);

  return 0;
}

