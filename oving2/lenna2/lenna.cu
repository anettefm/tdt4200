#include "hip/hip_runtime.h"
#include <iostream>
#include "lodepng.h"
#include <stdio.h>
#include <stdint.h> /* for uint64 definition */
#include <time.h> /* for clock_gettime */

 #define BILLION 1000000000L


__global__ void invert( unsigned char* image_d ){

        int id=blockIdx.x*blockDim.x+threadIdx.x;
        image_d[id] = ~image_d[id];
}


int main( int argc, char ** argv){
// variabler til tidtaking
	uint64_t diff;
	struct timespec start, end;

  size_t pngsize;
  unsigned char *png;
  const char * filename = "lenna512x512_inv.png";


        clock_gettime(CLOCK_MONOTONIC, &start); /* mark start time */

  /* Read in the image */
  lodepng_load_file(&png, &pngsize, filename);


clock_gettime(CLOCK_MONOTONIC, &end); /* mark the end time */
        diff = BILLION * (end.tv_sec - start.tv_sec) + end.tv_nsec - start.tv_nsec; printf("elapsed time = %llu nanoseconds\n", (long long unsigned int) diff);
 
  unsigned char *image;
  unsigned int width, height;
  /* Decode it into a RGB 8-bit per channel vector */
  unsigned int error = lodepng_decode24(&image, &width, &height, png, pngsize);

  /* Check if read and decode of .png went well */
  if(error != 0){
      std::cout << "error " << error << ": " << lodepng_error_text(error) << std::endl;
  }

  // Do work
         unsigned char* image_d;

        size_t size;
        size=height*width*3*sizeof(char);

        hipMalloc((void**) &image_d, size);

	clock_gettime(CLOCK_MONOTONIC, &start); /* mark start time */	

	hipMemcpy(image_d, image, size, hipMemcpyHostToDevice);
	hipDeviceSynchronize();	
	clock_gettime(CLOCK_MONOTONIC, &end); /* mark the end time */
        diff = BILLION * (end.tv_sec - start.tv_sec) + end.tv_nsec - start.tv_nsec; printf("elapsed time = %llu nanoseconds\n", (long long unsigned int) diff);

	int threadBlock=1024;
	int gridBlock=3*512*512/threadBlock;

        invert<<<gridBlock, threadBlock>>>(image_d);

        hipMemcpy(image, image_d, size, hipMemcpyDeviceToHost);

	clock_gettime(CLOCK_MONOTONIC, &end); /* mark the end time */
	diff = BILLION * (end.tv_sec - start.tv_sec) + end.tv_nsec - start.tv_nsec; printf("elapsed time = %llu nanoseconds\n", (long long unsigned int) diff);


  /* Save the result to a new .png file */
  lodepng_encode24_file("lenna512x512_orig.png", image , width,height);
  /*clean up */

        free(image);  hipFree(image_d);
  return 0;
}
