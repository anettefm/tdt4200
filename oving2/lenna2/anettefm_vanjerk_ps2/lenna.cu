#include "hip/hip_runtime.h"
#include <iostream>
#include "lodepng.h"
#include <stdio.h>
//#include <stdint.h> /* for uint64 definition */
//#include <time.h> /* for clock_gettime */

// #define BILLION 1000000000L


__global__ void invert( unsigned char* image_d ){

        int id=blockIdx.x*blockDim.x+threadIdx.x;
        image_d[id] = ~image_d[id];
}


int main( int argc, char ** argv){
// variabler til tidtaking
//	uint64_t diff;
//	struct timespec start_hele, start_minne, start_exe, end;

  size_t pngsize;
  unsigned char *png;
  const char * filename = "lenna512x512_inv.png";


 //       clock_gettime(CLOCK_MONOTONIC, &start_hele); /* mark start time */

  /* Read in the image */
  lodepng_load_file(&png, &pngsize, filename);

  unsigned char *image;
  unsigned int width, height;
  /* Decode it into a RGB 8-bit per channel vector */
  unsigned int error = lodepng_decode24(&image, &width, &height, png, pngsize);

  /* Check if read and decode of .png went well */
  if(error != 0){
      std::cout << "error " << error << ": " << lodepng_error_text(error) << std::endl;
  }

  // Do work
         unsigned char* image_d;

        size_t size;
        size=height*width*3*sizeof(char);

        hipMalloc((void**) &image_d, size);

//	clock_gettime(CLOCK_MONOTONIC, &start_minne); /* mark start time */	
//        clock_gettime(CLOCK_MONOTONIC, &start_exe); /* mark start time */

	hipMemcpy(image_d, image, size, hipMemcpyHostToDevice);
	hipDeviceSynchronize();	

//	clock_gettime(CLOCK_MONOTONIC, &end); /* mark the end time */
//        diff = BILLION * (end.tv_sec - start_minne.tv_sec) + end.tv_nsec - start_minne.tv_nsec; printf("minne: elapsed time = %llu nanoseconds\n", (long long unsigned int) diff);

	int threadBlock=1024;
	int gridBlock=3*512*512/threadBlock;

        invert<<<gridBlock, threadBlock>>>(image_d);

        hipMemcpy(image, image_d, size, hipMemcpyDeviceToHost);

//	clock_gettime(CLOCK_MONOTONIC, &end); /* mark the end time */
//	diff = BILLION * (end.tv_sec - start_exe.tv_sec) + end.tv_nsec - start_exe.tv_nsec; printf("exe: elapsed time = %llu nanoseconds\n", (long long unsigned int) diff);


  /* Save the result to a new .png file */
  lodepng_encode24_file("lenna512x512_orig.png", image , width,height);
  /*clean up */
//        clock_gettime(CLOCK_MONOTONIC, &end); /* mark the end time */
//        diff = BILLION * (end.tv_sec - start_hele.tv_sec) + end.tv_nsec - start_hele.tv_nsec; printf("hele: elapsed time = %llu nanoseconds\n", (long long unsigned int) diff);

        free(image);  hipFree(image_d);
  return 0;
}
