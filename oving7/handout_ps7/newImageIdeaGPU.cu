#include "hip/hip_runtime.h"
#include <math.h>
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include "ppmCU.h"

// Image from:
// http://7-themes.com/6971875-funny-flowers-pictures.html



// TODO: You should implement this
//__global__ void performNewIdeaFinalizationGPU( ... ) { ... }

// TODO: You should implement this
//__global__ void convertImageToNewFormatGPU( ... ) { ... }

// Perhaps some extra kernels will be practical as well?
//__global__ void ...GPU( ... ) { ... }

typedef struct {
     float red,green,blue;
} AccuratePixel;

typedef struct {
     int x, y;
     AccuratePixel *data;
} AccurateImage;

// Convert a PPM image to a high-precision format 
AccurateImage *convertImageToNewFormat(PPMImage *image) {
	// Make a copy
	AccurateImage *imageAccurate;
	imageAccurate = (AccurateImage *)malloc(sizeof(AccurateImage));
	imageAccurate->data = (AccuratePixel*)malloc(image->x * image->y * sizeof(AccuratePixel));
	for(int i = 0; i < image->x * image->y; i++) {
		imageAccurate->data[i].red   = (float)image->data[i].red;
		imageAccurate->data[i].green = (float)image->data[i].green;
		imageAccurate->data[i].blue  = (float)image->data[i].blue;
	}
	imageAccurate->x = image->x;
	imageAccurate->y = image->y;
	
	return imageAccurate;
}
// TODO: You should implement this
__global__ void convertImageToNewFormatGPU( unsigned char *imagePPM, AccuratePixel *image ) { 

	int senterX= blockIdx.x*blockDim.x+threadIdx.x;
	int senterY=blockIdx.y*blockDim.y+ threadIdx.y;

	int numberOfValuesInEachRow = gridDim.x*blockDim.x;
	int i = numberOfValuesInEachRow * senterY + senterX;

	image[i].red   =(float)(imagePPM[i].red);
	image[i].green = (float)(imagePPM[i].green);
	image[i].blue  =(float)(imagePPM[i].blue);
 }

// Convert a high-precision format to a PPM image
PPMImage *convertNewFormatToPPM(AccurateImage *image) {
	// Make a copy
	PPMImage *imagePPM;
	imagePPM = (PPMImage *)malloc(sizeof(PPMImage));
	imagePPM->data = (PPMPixel*)malloc(image->x * image->y * sizeof(PPMPixel));
	for(int i = 0; i < image->x * image->y; i++) {
		imagePPM->data[i].red   = (unsigned char) image->data[i].red;
		imagePPM->data[i].green = (unsigned char) image->data[i].green;
		imagePPM->data[i].blue  = (unsigned char) image->data[i].blue;
	}
	imagePPM->x = image->x;
	imagePPM->y = image->y;
	
	return imagePPM;
}

AccurateImage *createEmptyImage(PPMImage *image){
	AccurateImage *imageAccurate;
	imageAccurate = (AccurateImage *)malloc(sizeof(AccurateImage));
	imageAccurate->data = (AccuratePixel*)malloc(image->x * image->y * sizeof(AccuratePixel));
	imageAccurate->x = image->x;
	imageAccurate->y = image->y;
	
	return imageAccurate;
}

// free memory of an AccurateImage
void freeImage(AccurateImage *image){
	free(image->data);
	free(image);
}

/*void performNewIdeaIteration(AccurateImage *imageOut, AccurateImage *imageIn, int size) {
	
	// Iterate over each pixel
	for(int senterX = 0; senterX < imageIn->x; senterX++) {
	
		for(int senterY = 0; senterY < imageIn->y; senterY++) {
			
			// For each pixel we compute the magic number
			float sumR = 0;
			float sumG = 0;
			float sumB = 0;
			int countIncluded = 0;
			for(int x = -size; x <= size; x++) {
			
				for(int y = -size; y <= size; y++) {
					int currentX = senterX + x;
					int currentY = senterY + y;
					
					// Check if we are outside the bounds
					if(currentX < 0)
						continue;
					if(currentX >= imageIn->x)
						continue;
					if(currentY < 0)
						continue;
					if(currentY >= imageIn->y)
						continue;
					
					// Now we can begin
					int numberOfValuesInEachRow = imageIn->x; 
					int offsetOfThePixel = (numberOfValuesInEachRow * currentY + currentX);
					sumR += imageIn->data[offsetOfThePixel].red;
					sumG += imageIn->data[offsetOfThePixel].green;
					sumB += imageIn->data[offsetOfThePixel].blue;
				
					// Keep track of how many values we have included
					countIncluded++;
				}
			
			}
			
			// Now we compute the final value for all colours
			float valueR = sumR / countIncluded;
			float valueG = sumG / countIncluded;
			float valueB = sumB / countIncluded;
			
			// Update the output image
			int numberOfValuesInEachRow = imageOut->x; // R, G and B
			int offsetOfThePixel = (numberOfValuesInEachRow * senterY + senterX);
			imageOut->data[offsetOfThePixel].red = valueR;
			imageOut->data[offsetOfThePixel].green = valueG;
			imageOut->data[offsetOfThePixel].blue = valueB;
		}
	}
}*/
// TODO: You must implement this
// The handout code is much simpler than the MPI/OpenMP versions
__global__ void performNewIdeaIterationGPU(AccuratePixel *imageOut, AccuratePixel *imageIn, int size) { 

	int senterX= blockIdx.x*blockDim.x+threadIdx.x;
	int senterY=blockIdx.y*blockDim.y+ threadIdx.y;

	// For each pixel we compute the magic number
	float sumR = 0;
	float sumG = 0;
	float sumB = 0;
	int countIncluded = 0;
	for(int x = -size; x <= size; x++) {
	
		for(int y = -size; y <= size; y++) {
			int currentX = senterX + x;
			int currentY = senterY + y;
			
			// Check if we are outside the bounds
			if(currentX < 0)
				continue;
			if(currentX >=gridDim.x*blockDim.x)
				continue;
			if(currentY < 0)
				continue;
			if(currentY >= gridDim.y*blockDim.y)
				continue;
			
			// Now we can begin
			int numberOfValuesInEachRow = gridDim.x*blockDim.x; 
			int offsetOfThePixel = (numberOfValuesInEachRow * currentY + currentX);
			sumR += imageIn[offsetOfThePixel].red;
			sumG += imageIn[offsetOfThePixel].green;
			sumB +=imageIn[offsetOfThePixel].blue;
		
			// Keep track of how many values we have included
			countIncluded++;
		}
	
	}
	
	// Now we compute the final value for all colours
	float valueR = sumR / countIncluded;
	float valueG = sumG / countIncluded;
	float valueB = sumB / countIncluded;
	
	// Update the output image
	int numberOfValuesInEachRow =gridDim.x*blockDim.x; // R, G and B
	int offsetOfThePixel = (numberOfValuesInEachRow * senterY + senterX);
	imageOut[offsetOfThePixel].red = valueR;
	imageOut[offsetOfThePixel].green = valueG;
	imageOut[offsetOfThePixel].blue = valueB;

}

// Perform the final step, and save it as a ppm in imageOut
void performNewIdeaFinalization(AccurateImage *imageInSmall, AccurateImage *imageInLarge, PPMImage *imageOut) {

	
	imageOut->x = imageInSmall->x;
	imageOut->y = imageInSmall->y;
	
	for(int i = 0; i < imageInSmall->x * imageInSmall->y; i++) {
		float value = (imageInLarge->data[i].red - imageInSmall->data[i].red);
		if(value > 255.0f)
			imageOut->data[i].red = 255;
		else if (value < -1.0f) {
			value = 257.0f+value;
			if(value > 255.0f)
				imageOut->data[i].red = 255;
			else
				imageOut->data[i].red = floorf(value);
		} else if (value > -1.0f && value < 0.0f) {
			imageOut->data[i].red = 0;
		} else {
			imageOut->data[i].red = floorf(value);
		}
		
		value = (imageInLarge->data[i].green - imageInSmall->data[i].green);
		if(value > 255.0f)
			imageOut->data[i].green = 255;
		else if (value < -1.0f) {
			value = 257.0f+value;
			if(value > 255.0f)
				imageOut->data[i].green = 255;
			else
				imageOut->data[i].green = floorf(value);
		} else if (value > -1.0f && value < 0.0f) {
			imageOut->data[i].green = 0;
		} else {
			imageOut->data[i].green = floorf(value);
		}
		
		value = (imageInLarge->data[i].blue - imageInSmall->data[i].blue);
		if(value > 255.0f)
			imageOut->data[i].blue = 255;
		else if (value < -1.0f) {
			value = 257.0f+value;
			if(value > 255.0f)
				imageOut->data[i].blue = 255;
			else
				imageOut->data[i].blue = floorf(value);
		} else if (value > -1.0f && value < 0.0f) {
			imageOut->data[i].blue = 0;
		} else {
			imageOut->data[i].blue = floorf(value);
		}
	}
}

int main(int argc, char** argv) {
	
	PPMImage *image;
        
	if(argc > 1) {
		image = readPPM("flower.ppm");
	} else {
		image = readStreamPPM(stdin);
	}

//	AccurateImage *imageUnchanged = convertImageToNewFormat(image); // save the unchanged image from input image
	AccurateImage *imageSmall = createEmptyImage(image);
	AccurateImage *imageBig = createEmptyImage(image);

	
	PPMImage *imageOut;
	imageOut = (PPMImage *)malloc(sizeof(PPMImage));
	imageOut->data = (PPMPixel*)malloc(image->x * image->y * sizeof(PPMPixel));
	
	dim3 gridBlock, threadBlock;

	threadBlock.x=32;
	threadBlock.y=30;
    	gridBlock.x=image->x/threadBlock.x;
    	gridBlock.y=(image->y/threadBlock.y);

	hipError_t err = hipSuccess; 
	unsigned char *imagePPMGPU;
	hipMalloc((void**)& imagePPMGPU, 3*image->x*image->y*sizeof(char));
	err=hipMemcpy(imagePPMGPU, image->data, 3*image->x*image->y*sizeof(unsigned char), hipMemcpyHostToDevice);
    	if(err !=hipSuccess){
        	printf("%s-%d\n",hipGetErrorString(err),1);
        	getchar();  
    	}


	AccuratePixel *imageUnchangedGPU;
	AccuratePixel *imageSmallGPU;
	AccuratePixel *imageBigGPU;
	AccuratePixel *imageBufferGPU;

    err=hipMalloc((void**)&imageUnchangedGPU, 3*image->x * image->y * sizeof(float));
    err=hipMalloc((void**)&imageSmallGPU, 3*image->x * image->y * sizeof(float));
    err=hipMalloc((void**)&imageBigGPU, 3*image->x * image->y * sizeof(float));
    err=hipMalloc((void**)&imageBufferGPU, 3*image->x * image->y * sizeof(float));
    if(err !=0){
        printf("%s-%d\n",hipGetErrorString(err),1);
        getchar();  
    }

//	err=hipMemcpy(imageUnchangedGPU, imageUnchanged->data, 3*image->x*image->y*sizeof(float), hipMemcpyHostToDevice);
    	if(err !=0){
        	printf("%s-%d\n",hipGetErrorString(err),1);
        	getchar();  
    	}
	convertImageToNewFormatGPU<<<gridBlock, threadBlock>>>( imagePPMGPU, imageUnchangedGPU );


	// Process the tiny case:
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageSmallGPU, imageUnchangedGPU, 2);
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageBufferGPU, imageSmallGPU, 2);
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageSmallGPU, imageBufferGPU, 2);
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageBufferGPU, imageSmallGPU, 2);
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageSmallGPU, imageBufferGPU, 2);
	
	// Process the small case:
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageBigGPU, imageUnchangedGPU,3);
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageBufferGPU, imageBigGPU,3);
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageBigGPU, imageBufferGPU,3);
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageBufferGPU, imageBigGPU,3);
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageBigGPU, imageBufferGPU,3);
	
	err=hipMemcpy(imageSmall->data, imageSmallGPU, 3*image->x*image->y*sizeof(float), hipMemcpyDeviceToHost);
    if(err !=0){
        printf("%s-%d\n",hipGetErrorString(err),1);
        getchar();  
    }
	err=hipMemcpy(imageBig->data, imageBigGPU, 3*image->x*image->y*sizeof(float), hipMemcpyDeviceToHost);
    if(err !=0){
        printf("%s-%d\n",hipGetErrorString(err),1);
        getchar();  
    }

	// save tiny case result
	performNewIdeaFinalization(imageSmall,  imageBig, imageOut);
	if(argc > 1) {
		writePPM("flower_tiny.ppm", imageOut);
	} else {
		writeStreamPPM(stdout, imageOut);
	}

	
	// Process the medium case:
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageSmallGPU, imageUnchangedGPU, 5);
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageBufferGPU, imageSmallGPU, 5);
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageSmallGPU, imageBufferGPU, 5);
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageBufferGPU, imageSmallGPU, 5);
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageSmallGPU, imageBufferGPU, 5);

	err=hipMemcpy(imageSmall->data, imageSmallGPU, 3*image->x*image->y*sizeof(float), hipMemcpyDeviceToHost);
    if(err !=0){
        printf("%s-%d\n",hipGetErrorString(err),1);
        getchar();  
    }
	// save small case
	performNewIdeaFinalization(imageBig,  imageSmall,imageOut);
	if(argc > 1) {
		writePPM("flower_small.ppm", imageOut);
	} else {
		writeStreamPPM(stdout, imageOut);
	}

	// process the large case
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageBigGPU, imageUnchangedGPU, 8);
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageBufferGPU, imageBigGPU, 8);
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageBigGPU, imageBufferGPU, 8);
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageBufferGPU, imageBigGPU, 8);
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageBigGPU, imageBufferGPU, 8);

	err=hipMemcpy(imageBig->data, imageBigGPU, 3*image->x*image->y*sizeof(float), hipMemcpyDeviceToHost);
    if(err !=0){
        printf("%s-%d\n",hipGetErrorString(err),1);
        getchar();  
    }
	// save the medium case
	performNewIdeaFinalization(imageSmall,  imageBig, imageOut);
	if(argc > 1) {
		writePPM("flower_medium.ppm", imageOut);
	} else {
		writeStreamPPM(stdout, imageOut);
	}
	
	// free all memory structures
//	freeImage(imageUnchanged);
	freeImage(imageSmall);
	freeImage(imageBig);
	free(imageOut->data);
	free(imageOut);
	free(image->data);
	free(image);
	hipFree(imageBigGPU);
	hipFree(imageBufferGPU);
	hipFree(imageUnchangedGPU);
	hipFree(imageSmallGPU);
	return 0;
}

