#include "hip/hip_runtime.h"
#include <math.h>
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include "ppmCU.h"

// Image from:
// http://7-themes.com/6971875-funny-flowers-pictures.html



// TODO: You should implement this
//__global__ void performNewIdeaFinalizationGPU( ... ) { ... }

// TODO: You should implement this
//__global__ void convertImageToNewFormatGPU( ... ) { ... }

// Perhaps some extra kernels will be practical as well?
//__global__ void ...GPU( ... ) { ... }

typedef struct {
     float red,green,blue;
} AccuratePixel;

typedef struct {
     int x, y;
     AccuratePixel *data;
} AccurateImage;

// Convert a PPM image to a high-precision format 
AccurateImage *convertImageToNewFormat(PPMImage *image) {
	// Make a copy
	AccurateImage *imageAccurate;
	imageAccurate = (AccurateImage *)malloc(sizeof(AccurateImage));
	imageAccurate->data = (AccuratePixel*)malloc(image->x * image->y * sizeof(AccuratePixel));
	for(int i = 0; i < image->x * image->y; i++) {
		imageAccurate->data[i].red   = (float) image->data[i].red;
		imageAccurate->data[i].green = (float) image->data[i].green;
		imageAccurate->data[i].blue  = (float) image->data[i].blue;
	}
	imageAccurate->x = image->x;
	imageAccurate->y = image->y;
	
	return imageAccurate;
}

// Convert a high-precision format to a PPM image
PPMImage *convertNewFormatToPPM(AccurateImage *image) {
	// Make a copy
	PPMImage *imagePPM;
	imagePPM = (PPMImage *)malloc(sizeof(PPMImage));
	imagePPM->data = (PPMPixel*)malloc(image->x * image->y * sizeof(PPMPixel));
	for(int i = 0; i < image->x * image->y; i++) {
		imagePPM->data[i].red   = (unsigned char) image->data[i].red;
		imagePPM->data[i].green = (unsigned char) image->data[i].green;
		imagePPM->data[i].blue  = (unsigned char) image->data[i].blue;
	}
	imagePPM->x = image->x;
	imagePPM->y = image->y;
	
	return imagePPM;
}

AccurateImage *createEmptyImage(PPMImage *image){
	AccurateImage *imageAccurate;
	imageAccurate = (AccurateImage *)malloc(sizeof(AccurateImage));
	imageAccurate->data = (AccuratePixel*)malloc(image->x * image->y * sizeof(AccuratePixel));
	imageAccurate->x = image->x;
	imageAccurate->y = image->y;
	
	return imageAccurate;
}

// free memory of an AccurateImage
void freeImage(AccurateImage *image){
	free(image->data);
	free(image);
}

/*void performNewIdeaIteration(AccurateImage *imageOut, AccurateImage *imageIn, int size) {
	
	// Iterate over each pixel
	for(int senterX = 0; senterX < imageIn->x; senterX++) {
	
		for(int senterY = 0; senterY < imageIn->y; senterY++) {
			
			// For each pixel we compute the magic number
			float sumR = 0;
			float sumG = 0;
			float sumB = 0;
			int countIncluded = 0;
			for(int x = -size; x <= size; x++) {
			
				for(int y = -size; y <= size; y++) {
					int currentX = senterX + x;
					int currentY = senterY + y;
					
					// Check if we are outside the bounds
					if(currentX < 0)
						continue;
					if(currentX >= imageIn->x)
						continue;
					if(currentY < 0)
						continue;
					if(currentY >= imageIn->y)
						continue;
					
					// Now we can begin
					int numberOfValuesInEachRow = imageIn->x; 
					int offsetOfThePixel = (numberOfValuesInEachRow * currentY + currentX);
					sumR += imageIn->data[offsetOfThePixel].red;
					sumG += imageIn->data[offsetOfThePixel].green;
					sumB += imageIn->data[offsetOfThePixel].blue;
				
					// Keep track of how many values we have included
					countIncluded++;
				}
			
			}
			
			// Now we compute the final value for all colours
			float valueR = sumR / countIncluded;
			float valueG = sumG / countIncluded;
			float valueB = sumB / countIncluded;
			
			// Update the output image
			int numberOfValuesInEachRow = imageOut->x; // R, G and B
			int offsetOfThePixel = (numberOfValuesInEachRow * senterY + senterX);
			imageOut->data[offsetOfThePixel].red = valueR;
			imageOut->data[offsetOfThePixel].green = valueG;
			imageOut->data[offsetOfThePixel].blue = valueB;
		}
	}
}*/
// TODO: You must implement this
// The handout code is much simpler than the MPI/OpenMP versions
__global__ void performNewIdeaIterationGPU(AccurateImage *imageOut, AccurateImage *imageIn, int size) { 

	int senterX=threadIdx.x;
	int senterY=threadIdx.y;

	// For each pixel we compute the magic number
	float sumR = 0;
	float sumG = 0;
	float sumB = 0;
	int countIncluded = 0;
	for(int x = -size; x <= size; x++) {
	
		for(int y = -size; y <= size; y++) {
			int currentX = senterX + x;
			int currentY = senterY + y;
			
			// Check if we are outside the bounds
			if(currentX < 0)
				continue;
			if(currentX >= blockDim.x)
				continue;
			if(currentY < 0)
				continue;
			if(currentY >= blockDim.y)
				continue;
			
			// Now we can begin
			int numberOfValuesInEachRow = blockDim.y; 
			int offsetOfThePixel = (numberOfValuesInEachRow * currentY + currentX);
			sumR += imageIn->data[offsetOfThePixel].red;
			sumG += imageIn->data[offsetOfThePixel].green;
			sumB += imageIn->data[offsetOfThePixel].blue;
		
			// Keep track of how many values we have included
			countIncluded++;
		}
	
	}
	
	// Now we compute the final value for all colours
	float valueR = sumR / countIncluded;
	float valueG = sumG / countIncluded;
	float valueB = sumB / countIncluded;
	
	// Update the output image
	int numberOfValuesInEachRow = blockDim.x; // R, G and B
	int offsetOfThePixel = (numberOfValuesInEachRow * senterY + senterX);
	imageOut->data[offsetOfThePixel].red = valueR;
	imageOut->data[offsetOfThePixel].green = valueG;
	imageOut->data[offsetOfThePixel].blue = valueB;
}

// Perform the final step, and save it as a ppm in imageOut
void performNewIdeaFinalization(AccurateImage *imageInSmall, AccurateImage *imageInLarge, PPMImage *imageOut) {

	
	imageOut->x = imageInSmall->x;
	imageOut->y = imageInSmall->y;
	
	for(int i = 0; i < imageInSmall->x * imageInSmall->y; i++) {
		float value = (imageInLarge->data[i].red - imageInSmall->data[i].red);
		if(value > 255.0f)
			imageOut->data[i].red = 255;
		else if (value < -1.0f) {
			value = 257.0f+value;
			if(value > 255.0f)
				imageOut->data[i].red = 255;
			else
				imageOut->data[i].red = floorf(value);
		} else if (value > -1.0f && value < 0.0f) {
			imageOut->data[i].red = 0;
		} else {
			imageOut->data[i].red = floorf(value);
		}
		
		value = (imageInLarge->data[i].green - imageInSmall->data[i].green);
		if(value > 255.0f)
			imageOut->data[i].green = 255;
		else if (value < -1.0f) {
			value = 257.0f+value;
			if(value > 255.0f)
				imageOut->data[i].green = 255;
			else
				imageOut->data[i].green = floorf(value);
		} else if (value > -1.0f && value < 0.0f) {
			imageOut->data[i].green = 0;
		} else {
			imageOut->data[i].green = floorf(value);
		}
		
		value = (imageInLarge->data[i].blue - imageInSmall->data[i].blue);
		if(value > 255.0f)
			imageOut->data[i].blue = 255;
		else if (value < -1.0f) {
			value = 257.0f+value;
			if(value > 255.0f)
				imageOut->data[i].blue = 255;
			else
				imageOut->data[i].blue = floorf(value);
		} else if (value > -1.0f && value < 0.0f) {
			imageOut->data[i].blue = 0;
		} else {
			imageOut->data[i].blue = floorf(value);
		}
	}
}

int main(int argc, char** argv) {
	
	PPMImage *image;
        
	if(argc > 1) {
		image = readPPM("flower.ppm");
	} else {
		image = readStreamPPM(stdin);
	}

	AccurateImage *imageUnchanged = convertImageToNewFormat(image); // save the unchanged image from input image
	AccurateImage *imageBuffer = createEmptyImage(image);
	AccurateImage *imageSmall = createEmptyImage(image);
	AccurateImage *imageBig = createEmptyImage(image);
	
	PPMImage *imageOut;
	imageOut = (PPMImage *)malloc(sizeof(PPMImage));
	imageOut->data = (PPMPixel*)malloc(image->x * image->y * sizeof(PPMPixel));
	
	dim3 gridBlock, threadBlock;

	threadBlock.x=32;
	threadBlock.y=30;
    gridBlock.x=imageBuffer->x/threadBlock.x;
    gridBlock.y=imageBuffer->y/threadBlock.y;

/*
    int numPoints    = 16,
        gpuBlockSize = 4,
        pointSize    = sizeof(point),
        numBytes     = numPoints * pointSize,
        gpuGridSize  = numPoints / gpuBlockSize;

        // allocate memory
    point *cpuPointArray,
          *gpuPointArray;
    cpuPointArray = (point*)malloc(numBytes);
    hipMalloc((void**)&gpuPointArray, numBytes);*/

	AccurateImage *imageUnchangedGPU;
	AccurateImage *imageSmallGPU;
	AccurateImage *imageBigGPU;
	AccurateImage *imageBufferGPU;

    hipMalloc((void**)&imageUnchangedGPU, image->x * image->y * sizeof(AccuratePixel));
    hipMalloc((void**)&imageSmallGPU, image->x * image->y * sizeof(AccuratePixel));
    hipMalloc((void**)&imageBigGPU, image->x * image->y * sizeof(AccuratePixel));
    hipMalloc((void**)&imageBufferGPU, image->x * image->y * sizeof(AccuratePixel));

    hipMemcpy(imageUnchangedGPU->data, imageUnchanged->data, 3*imageUnchanged->x*imageUnchanged->y, hipMemcpyHostToDevice);


	// Process the tiny case:
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageSmallGPU, imageUnchangedGPU, 2);
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageBufferGPU, imageSmallGPU, 2);
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageSmallGPU, imageBufferGPU, 2);
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageBufferGPU, imageSmallGPU, 2);
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageSmallGPU, imageBufferGPU, 2);
	
	// Process the small case:
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageBigGPU, imageUnchangedGPU,3);
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageBufferGPU, imageBigGPU,3);
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageBigGPU, imageBufferGPU,3);
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageBufferGPU, imageBigGPU,3);
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageBigGPU, imageBufferGPU,3);
	
	hipMemcpy(imageSmall->data, imageSmallGPU->data, 3*imageUnchanged->x*imageUnchanged->y, hipMemcpyDeviceToHost);
	hipMemcpy(imageBig->data, imageBigGPU->data, 3*imageUnchanged->x*imageUnchanged->y, hipMemcpyDeviceToHost);
	// save tiny case result
	performNewIdeaFinalization(imageSmall,  imageBig, imageOut);
	if(argc > 1) {
		writePPM("flower_tinyGPU.ppm", imageOut);
	} else {
		writeStreamPPM(stdout, imageOut);
	}

	
	// Process the medium case:
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageSmallGPU, imageUnchangedGPU, 5);
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageBufferGPU, imageSmallGPU, 5);
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageSmallGPU, imageBufferGPU, 5);
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageBufferGPU, imageSmallGPU, 5);
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageSmallGPU, imageBufferGPU, 5);

	hipMemcpy(imageSmall->data, imageSmallGPU->data, 3*imageUnchanged->x*imageUnchanged->y, hipMemcpyDeviceToHost);

	// save small case
	performNewIdeaFinalization(imageBig,  imageSmall,imageOut);
	if(argc > 1) {
		writePPM("flower_smallGPU.ppm", imageOut);
	} else {
		writeStreamPPM(stdout, imageOut);
	}

	// process the large case
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageBigGPU, imageUnchangedGPU, 8);
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageBufferGPU, imageBigGPU, 8);
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageBigGPU, imageBufferGPU, 8);
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageBufferGPU, imageBigGPU, 8);
	performNewIdeaIterationGPU<<<gridBlock, threadBlock>>>(imageBigGPU, imageBufferGPU, 8);

	hipMemcpy(imageBig->data, imageBigGPU->data, 3*imageUnchanged->x*imageUnchanged->y, hipMemcpyDeviceToHost);

	// save the medium case
	performNewIdeaFinalization(imageSmall,  imageBig, imageOut);
	if(argc > 1) {
		writePPM("flower_mediumGPU.ppm", imageOut);
	} else {
		writeStreamPPM(stdout, imageOut);
	}
	
	// free all memory structures
	freeImage(imageUnchanged);
	freeImage(imageBuffer);
	freeImage(imageSmall);
	freeImage(imageBig);
	free(imageOut->data);
	free(imageOut);
	free(image->data);
	free(image);
	
	return 0;
}

